
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__  void square(float *d_out, float *d_in) {
    int idx = threadIdx.x;
    float f = d_in[idx];
    d_out[idx] = f * f; 
}

int main(int argc, char **argv) {
    const int ARRAY_SIZE =  1024;
    const int ARRRAY_BYTES = ARRAY_SIZE * sizeof(float);

    float h_in[ARRAY_SIZE];

    for (int i = 0; i < ARRAY_SIZE; i++) {
        h_in[i] = (float) i;
    }

    float h_out[ARRAY_SIZE];

    float *d_in;
    float *d_out;

    hipMalloc((void **) &d_in, ARRRAY_BYTES);
    hipMalloc((void **) &d_out, ARRRAY_BYTES);

    hipMemcpy(d_in, h_in, ARRRAY_BYTES, hipMemcpyHostToDevice);

    square<<<1, ARRAY_SIZE>>>(d_out, d_in);

    hipMemcpy(h_out, d_out, ARRRAY_BYTES, hipMemcpyDeviceToHost);

    for (int i = 0; i < ARRAY_SIZE; i++) {
        printf("%f", h_out[i]);
        printf(((i % 4) != 3) ? "\t" : "\n");
    }

    hipFree(d_in);
    hipFree(d_out);

    return 0;
}