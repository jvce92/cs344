#include "hip/hip_runtime.h"
#include <stdio.h>
#include "gputimer.h"
#include <string>

#define NUM_THREADS 10000000
#define ARRAY_SIZE  100

#define BLOCK_WIDTH 1000

void print_array(int *array, int size)
{
    printf("{ ");
    for (int i = 0; i < size; i++)  { printf("%d ", array[i]); }
    printf("}\n");
}

__global__ void increment_naive(int *g)
{
	// which thread is this?
	int i = blockIdx.x * blockDim.x + threadIdx.x; 

	// each thread to increment consecutive elements, wrapping at ARRAY_SIZE
	i = i % ARRAY_SIZE;  
	g[i] = g[i] + 1;
}

__global__ void increment_atomic(int *g)
{
	// which thread is this?
	int i = blockIdx.x * blockDim.x + threadIdx.x; 

	// each thread to increment consecutive elements, wrapping at ARRAY_SIZE
	i = i % ARRAY_SIZE;  
	atomicAdd(& g[i], 1);
}

int main(int argc,char **argv)
{   
    if (argc < 2) return 1;
    std::string mode = std::string(argv[1]);

    GpuTimer timer;
    printf("%d total threads in %d blocks writing into %d array elements\n",
           NUM_THREADS, NUM_THREADS / BLOCK_WIDTH, ARRAY_SIZE);

    // declare and allocate host memory
    int h_array[ARRAY_SIZE];
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(int);
 
    // declare, allocate, and zero out GPU memory
    int * d_array;
    hipMalloc((void **) &d_array, ARRAY_BYTES);
    hipMemset((void *) d_array, 0, ARRAY_BYTES); 

    // launch the kernel - comment out one of these
    timer.Start();
    
    if (mode.compare("naive") == 0) {
        increment_naive<<<NUM_THREADS/BLOCK_WIDTH, BLOCK_WIDTH>>>(d_array);
    }
    else if (mode.compare("atomic") == 0) {
        increment_atomic<<<NUM_THREADS/BLOCK_WIDTH, BLOCK_WIDTH>>>(d_array);
    }
    else {
        printf("The argument must be either naive or atomic\n");
        return 1;
    }

    timer.Stop();
    
    // copy back the array of sums from GPU and print
    hipMemcpy(h_array, d_array, ARRAY_BYTES, hipMemcpyDeviceToHost);
    print_array(h_array, ARRAY_SIZE);
    printf("Time elapsed = %g ms\n", timer.Elapsed());
 
    // free GPU memory allocation and exit
    hipFree(d_array);
    return 0;
}